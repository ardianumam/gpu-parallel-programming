#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

__global__ void vectorAddKernel(float* A, float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int N = 1000000; // Size of vectors
    size_t size = N * sizeof(float);

    // Host vectors
    float *h_A = new float[N];
    float *h_B = new float[N];
    float *h_C = new float[N];

    // Initialize host vectors
    for (int i = 0; i < N; i++) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i * 2);
    }

    // Device vectors
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Create CUDA events for time measurement
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start, 0);

    // Transfer data to GPU
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Kernel configuration
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernel on the device
    vectorAddKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Transfer the result back to the host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Record the stop event
    hipEventRecord(stop, 0);

    // Synchronize to make sure all operations are completed
    hipEventSynchronize(stop);

    // Calculate the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Clean up
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    std::cout << "Vector addition completed successfully." << std::endl;
    std::cout << "Elapsed time: " << milliseconds << " ms" << std::endl;

    return 0;
}
