#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vector_addition(int *A, int *B, int *C, int size) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < size)
        C[tid] = A[tid] + B[tid];
}

float single_stream(int *A_Host, int *B_Host, int *C_Host, int size, int DimBlock, int DimGrid) {
    int *A_GPU, *B_GPU, *C_GPU;
    hipMalloc(&A_GPU, sizeof(int) * size);
    hipMalloc(&B_GPU, sizeof(int) * size);
    hipMalloc(&C_GPU, sizeof(int) * size);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    hipMemcpy(A_GPU, A_Host, sizeof(int) * size, hipMemcpyHostToDevice);
    hipMemcpy(B_GPU, B_Host, sizeof(int) * size, hipMemcpyHostToDevice);
    vector_addition<<<DimGrid, DimBlock>>>(A_GPU, B_GPU, C_GPU, size);
    hipMemcpy(C_Host, C_GPU, sizeof(int) * size, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);

    hipFree(A_GPU);
    hipFree(B_GPU);
    hipFree(C_GPU);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return ms;
}

float two_streams_data_transfer(int *A_Host, int *B_Host, int *C_Host, int size, int DimBlock, int DimGrid) {
    int *A_GPU, *B_GPU, *C_GPU;
    hipMalloc(&A_GPU, sizeof(int) * size);
    hipMalloc(&B_GPU, sizeof(int) * size);
    hipMalloc(&C_GPU, sizeof(int) * size);

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    hipEvent_t start, stop, event1, event2;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&event1);
    hipEventCreate(&event2);

    hipEventRecord(start);
    hipMemcpyAsync(A_GPU, A_Host, sizeof(int) * size, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(B_GPU, B_Host, sizeof(int) * size, hipMemcpyHostToDevice, stream2);
    hipEventRecord(event1, stream1);
    hipEventRecord(event2, stream2);
    hipStreamWaitEvent(0, event1, 0);
    hipStreamWaitEvent(0, event2, 0);
    
    vector_addition<<<DimGrid, DimBlock>>>(A_GPU, B_GPU, C_GPU, size);
    hipMemcpy(C_Host, C_GPU, sizeof(int) * size, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);

    hipFree(A_GPU);
    hipFree(B_GPU);
    hipFree(C_GPU);

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(event1);
    hipEventDestroy(event2);

    return ms;
}

float two_streams_full_pipeline(int *A_Host, int *B_Host, int *C_Host, int size, int DimBlock, int DimGrid) {
    int halfSize = size / 2;
    int *A1_GPU, *B1_GPU, *C1_GPU, *A2_GPU, *B2_GPU, *C2_GPU;
    hipMalloc(&A1_GPU, sizeof(int) * halfSize);
    hipMalloc(&B1_GPU, sizeof(int) * halfSize);
    hipMalloc(&C1_GPU, sizeof(int) * halfSize);
    hipMalloc(&A2_GPU, sizeof(int) * halfSize);
    hipMalloc(&B2_GPU, sizeof(int) * halfSize);
    hipMalloc(&C2_GPU, sizeof(int) * halfSize);

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    hipMemcpyAsync(A1_GPU, A_Host, sizeof(int) * halfSize, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(B1_GPU, B_Host, sizeof(int) * halfSize, hipMemcpyHostToDevice, stream1);
    vector_addition<<<DimGrid, DimBlock, 0, stream1>>>(A1_GPU, B1_GPU, C1_GPU, halfSize);
    hipMemcpyAsync(C_Host, C1_GPU, sizeof(int) * halfSize, hipMemcpyDeviceToHost, stream1);

    hipMemcpyAsync(A2_GPU, A_Host + halfSize, sizeof(int) * halfSize, hipMemcpyHostToDevice, stream2);
    hipMemcpyAsync(B2_GPU, B_Host + halfSize, sizeof(int) * halfSize, hipMemcpyHostToDevice, stream2);
    vector_addition<<<DimGrid, DimBlock, 0, stream2>>>(A2_GPU, B2_GPU, C2_GPU, halfSize);
    hipMemcpyAsync(C_Host + halfSize, C2_GPU, sizeof(int) * halfSize, hipMemcpyDeviceToHost, stream2);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);

    hipFree(A1_GPU);
    hipFree(B1_GPU);
    hipFree(C1_GPU);
    hipFree(A2_GPU);
    hipFree(B2_GPU);
    hipFree(C2_GPU);

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return ms;
}

float three_streams_full_pipeline(int *A_Host, int *B_Host, int *C_Host, int size, int DimBlock, int DimGrid) {
    int thirdSize = size / 3;
    int *A1_GPU, *B1_GPU, *C1_GPU, *A2_GPU, *B2_GPU, *C2_GPU, *A3_GPU, *B3_GPU, *C3_GPU;
    hipMalloc(&A1_GPU, sizeof(int) * thirdSize);
    hipMalloc(&B1_GPU, sizeof(int) * thirdSize);
    hipMalloc(&C1_GPU, sizeof(int) * thirdSize);
    hipMalloc(&A2_GPU, sizeof(int) * thirdSize);
    hipMalloc(&B2_GPU, sizeof(int) * thirdSize);
    hipMalloc(&C2_GPU, sizeof(int) * thirdSize);
    hipMalloc(&A3_GPU, sizeof(int) * thirdSize);
    hipMalloc(&B3_GPU, sizeof(int) * thirdSize);
    hipMalloc(&C3_GPU, sizeof(int) * thirdSize);

    hipStream_t stream1, stream2, stream3;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    hipMemcpyAsync(A1_GPU, A_Host, sizeof(int) * thirdSize, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(B1_GPU, B_Host, sizeof(int) * thirdSize, hipMemcpyHostToDevice, stream1);
    vector_addition<<<DimGrid, DimBlock, 0, stream1>>>(A1_GPU, B1_GPU, C1_GPU, thirdSize);
    hipMemcpyAsync(C_Host, C1_GPU, sizeof(int) * thirdSize, hipMemcpyDeviceToHost, stream1);

    hipMemcpyAsync(A2_GPU, A_Host + thirdSize, sizeof(int) * thirdSize, hipMemcpyHostToDevice, stream2);
    hipMemcpyAsync(B2_GPU, B_Host + thirdSize, sizeof(int) * thirdSize, hipMemcpyHostToDevice, stream2);
    vector_addition<<<DimGrid, DimBlock, 0, stream2>>>(A2_GPU, B2_GPU, C2_GPU, thirdSize);
    hipMemcpyAsync(C_Host + thirdSize, C2_GPU, sizeof(int) * thirdSize, hipMemcpyDeviceToHost, stream2);

    hipMemcpyAsync(A3_GPU, A_Host + 2 * thirdSize, sizeof(int) * thirdSize, hipMemcpyHostToDevice, stream3);
    hipMemcpyAsync(B3_GPU, B_Host + 2 * thirdSize, sizeof(int) * thirdSize, hipMemcpyHostToDevice, stream3);
    vector_addition<<<DimGrid, DimBlock, 0, stream3>>>(A3_GPU, B3_GPU, C3_GPU, thirdSize);
    hipMemcpyAsync(C_Host + 2 * thirdSize, C3_GPU, sizeof(int) * thirdSize, hipMemcpyDeviceToHost, stream3);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);

    hipFree(A1_GPU);
    hipFree(B1_GPU);
    hipFree(C1_GPU);
    hipFree(A2_GPU);
    hipFree(B2_GPU);
    hipFree(C2_GPU);

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return ms;
}

int main() {
    int size = 1000002; // The size of the arrays
    int ThreadPerBlock = 256; // The size of the blocks
    int BlockPerGrid; 
    int *A_Host, *B_Host, *C_Host;


    hipHostMalloc((void**)&A_Host, sizeof(int) * size, hipHostMallocDefault);
    hipHostMalloc((void**)&B_Host, sizeof(int) * size, hipHostMallocDefault);
    hipHostMalloc((void**)&C_Host, sizeof(int) * size, hipHostMallocDefault);
    
    for (int i = 0; i < size; i++) {
        A_Host[i] = i + 1;
        B_Host[i] = 0;
    }

    int iterations = 50;
    float totalSingleStreamTime = 0;
    float totalTwoStreamDataTransferTime = 0;
    float totalTwoStreamFullPipelineTime = 0;
    float totalThreeStreamFullPipelineTime = 0;

    for (int i = 0; i < iterations; i++) {
        BlockPerGrid = ((size / 1) + ThreadPerBlock - 1) / ThreadPerBlock;
        totalSingleStreamTime += single_stream(A_Host, B_Host, C_Host, size, ThreadPerBlock, BlockPerGrid);

        BlockPerGrid = ((size / 1) + ThreadPerBlock - 1) / ThreadPerBlock;
        totalTwoStreamDataTransferTime += two_streams_data_transfer(A_Host, B_Host, C_Host, size, ThreadPerBlock, BlockPerGrid);
        
        BlockPerGrid = ((size / 2) + ThreadPerBlock - 1) / ThreadPerBlock;
        totalTwoStreamFullPipelineTime += two_streams_full_pipeline(A_Host, B_Host, C_Host, size, ThreadPerBlock, BlockPerGrid);
        
        BlockPerGrid = ((size / 3) + ThreadPerBlock - 1) / ThreadPerBlock;
        totalThreeStreamFullPipelineTime += three_streams_full_pipeline(A_Host, B_Host, C_Host, size, ThreadPerBlock, BlockPerGrid);
    }

    float averageSingleStreamTime = totalSingleStreamTime / iterations;
    float averageTwoStreamDataTransferTime = totalTwoStreamDataTransferTime / iterations;
    float averageTwoStreamFullPipelineTime = totalTwoStreamFullPipelineTime / iterations;
    float averageThreeStreamFullPipelineTime = totalThreeStreamFullPipelineTime / iterations;

    printf("Average Time (Single Stream): %f ms\n", averageSingleStreamTime);
    printf("Average Time (Two Streams for Data Transfer): %f ms\n", averageTwoStreamDataTransferTime);
    printf("Average Time (Two Streams Full Pipeline): %f ms\n", averageTwoStreamFullPipelineTime);
    printf("Average Time (Three Streams Full Pipeline): %f ms\n", averageThreeStreamFullPipelineTime);

    hipHostFree(A_Host);
    hipHostFree(B_Host);
    hipHostFree(C_Host);

    return 0;
}
