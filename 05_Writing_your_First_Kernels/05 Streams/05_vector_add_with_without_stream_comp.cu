#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

__global__ void vectorAddKernel(float* A, float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

float runSingleStream(int N, size_t size, float* h_A, float* h_B, float* h_C, float* d_A, float* d_B, float* d_C) {
    // Create CUDA events for time measurement
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start, 0);

    // Transfer data to GPU
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Kernel configuration
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernel on the device
    vectorAddKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Transfer the result back to the host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Record the stop event
    hipEventRecord(stop, 0);

    // Synchronize to make sure all operations are completed
    hipEventSynchronize(stop);

    // Calculate the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Clean up
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return milliseconds;
}

float runMultiStream(int N, size_t size, float* h_A, float* h_B, float* h_C, float* d_A, float* d_B, float* d_C) {
    // Create CUDA streams
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    // Create CUDA events for time measurement
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start, 0);

    // Transfer data to GPU asynchronously
    hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(d_B, h_B, size, hipMemcpyHostToDevice, stream2);

    // Create an event to synchronize streams
    hipEvent_t event;
    hipEventCreate(&event);

    // Record an event in stream2 after the data transfer
    hipEventRecord(event, stream2);

    // Wait for the event in stream1 before launching the kernel
    hipStreamWaitEvent(stream1, event, 0);

    // Kernel configuration
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernel on the device asynchronously
    vectorAddKernel<<<blocksPerGrid, threadsPerBlock, 0, stream1>>>(d_A, d_B, d_C, N);

    // Transfer the result back to the host asynchronously
    hipMemcpyAsync(h_C, d_C, size, hipMemcpyDeviceToHost, stream1);

    // Record the stop event
    hipEventRecord(stop, 0);

    // Synchronize the streams
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    // Calculate the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Clean up
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(event);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    return milliseconds;
}

int main() {
    int N = 1000000; // Size of vectors
    size_t size = N * sizeof(float);

    // Host vectors
    float *h_A = new float[N];
    float *h_B = new float[N];
    float *h_C = new float[N];

    // Initialize host vectors
    for (int i = 0; i < N; i++) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i * 2);
    }

    // Device vectors
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    int iterations = 50;
    float totalSingleStreamTime = 0;
    float totalMultiStreamTime = 0;

    for (int i = 0; i < iterations; i++) {
        totalSingleStreamTime += runSingleStream(N, size, h_A, h_B, h_C, d_A, d_B, d_C);
        totalMultiStreamTime += runMultiStream(N, size, h_A, h_B, h_C, d_A, d_B, d_C);
    }

    float averageSingleStreamTime = totalSingleStreamTime / iterations;
    float averageMultiStreamTime = totalMultiStreamTime / iterations;

    // Clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    std::cout << "Average elapsed time (Single Stream): " << averageSingleStreamTime << " ms" << std::endl;
    std::cout << "Average elapsed time (Multi Stream): " << averageMultiStreamTime << " ms" << std::endl;

    return 0;
}
