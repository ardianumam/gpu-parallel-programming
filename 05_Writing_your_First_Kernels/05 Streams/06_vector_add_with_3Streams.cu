
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vector_addition(int *A,int *B,int *C,int size)//CUDA kernel
{
	int tid = blockDim.x*blockIdx.x+threadIdx.x;//Global thread id
	if(tid<size)
		C[tid] = A[tid] + B[tid];//Vector Addition performs
}

int main()
{
	int size = 10000002;//The size of the arrays  (is the multiples of 3)
	int ThreadPerBlock = 1024;//The size of the blocks (The maximum value that we can set (hardware limitation) )
	int BlockPerGrid = ((size/3)-1)/ThreadPerBlock+1;//The number of blocks
	int *A_Host,*B_Host,*C_Host;

	hipHostMalloc((void**)&A_Host, sizeof(int)*size, hipHostMallocDefault);//is allocated in the heap region of the memory on the CPU (Pinned)
	hipHostMalloc((void**)&B_Host, sizeof(int)*size, hipHostMallocDefault);//is allocated in the heap region of the memory on the CPU (Pinned)
	hipHostMalloc((void**)&C_Host, sizeof(int)*size, hipHostMallocDefault);//is allocated in the heap region of the memory on the CPU (Pinned)
	
	for(int i=1;i<=size;i++)//The values are assigned to the arrays
	{
		A_Host[i-1] = i;
		B_Host[i-1] = 0;
	}

	int *A1_GPU,*B1_GPU,*C1_GPU;
	int *A2_GPU,*B2_GPU,*C2_GPU;
	int *A3_GPU,*B3_GPU,*C3_GPU;
	hipMalloc(&A1_GPU,sizeof(int)*size/3);//is allocated on the global memory of the GPU
	hipMalloc(&B1_GPU,sizeof(int)*size/3);//is allocated on the global memory of the GPU
	hipMalloc(&C1_GPU,sizeof(int)*size/3);//is allocated on the global memory of the GPU
	hipMalloc(&A2_GPU,sizeof(int)*size/3);//is allocated on the global memory of the GPU
	hipMalloc(&B2_GPU,sizeof(int)*size/3);//is allocated on the global memory of the GPU
	hipMalloc(&C2_GPU,sizeof(int)*size/3);//is allocated on the global memory of the GPU
	hipMalloc(&A3_GPU,sizeof(int)*size/3);//is allocated on the global memory of the GPU
	hipMalloc(&B3_GPU,sizeof(int)*size/3);//is allocated on the global memory of the GPU
	hipMalloc(&C3_GPU,sizeof(int)*size/3);//is allocated on the global memory of the GPU

	dim3 DimBlock(ThreadPerBlock);//The number of threads in a block
	dim3 DimGrid(BlockPerGrid);//The number of blocks in the grid

	hipEvent_t start, stop;//Variables for the time
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float totaltime;

	hipStream_t stream[3];
	hipStreamCreate(&stream[0]);//First stream is created
	hipStreamCreate(&stream[1]);//Second stream is created
	hipStreamCreate(&stream[2]);//Third stream is created

	hipEventRecord(start);//Time is started

  //The operations of each part are overlapped by using CUDA Streams
	hipMemcpyAsync(A1_GPU,A_Host+0*(size/3),sizeof(int)*size/3,hipMemcpyHostToDevice,stream[0]);//Copying data from CPU to GPU
	hipMemcpyAsync(B1_GPU,B_Host+0*(size/3),sizeof(int)*size/3,hipMemcpyHostToDevice,stream[0]);//Copying data from CPU to GPU
	vector_addition<<<DimGrid,DimBlock,0,stream[0]>>>(A1_GPU,B1_GPU,C1_GPU,size/3);//CUDA kernel is executed
	hipMemcpyAsync(C_Host+0*(size/3),C1_GPU,sizeof(int)*size/3,hipMemcpyDeviceToHost,stream[0]);//Copying data from GPU to CPU

	hipMemcpyAsync(A2_GPU,A_Host+1*(size/3),sizeof(int)*size/3,hipMemcpyHostToDevice,stream[1]);//Copying data from CPU to GPU
	hipMemcpyAsync(B2_GPU,B_Host+1*(size/3),sizeof(int)*size/3,hipMemcpyHostToDevice,stream[1]);//Copying data from CPU to GPU
	vector_addition<<<DimGrid,DimBlock,0,stream[1]>>>(A2_GPU,B2_GPU,C2_GPU,size/3);//CUDA kernel is executed
	hipMemcpyAsync(C_Host+1*(size/3),C2_GPU,sizeof(int)*size/3,hipMemcpyDeviceToHost,stream[1]);//Copying data from GPU to CPU

	hipMemcpyAsync(A3_GPU,A_Host+2*(size/3),sizeof(int)*size/3,hipMemcpyHostToDevice,stream[2]);//Copying data from CPU to GPU
	hipMemcpyAsync(B3_GPU,B_Host+2*(size/3),sizeof(int)*size/3,hipMemcpyHostToDevice,stream[2]);//Copying data from CPU to GPU
	vector_addition<<<DimGrid,DimBlock,0,stream[2]>>>(A3_GPU,B3_GPU,C3_GPU,size/3);//CUDA kernel is executed
	hipMemcpyAsync(C_Host+2*(size/3),C3_GPU,sizeof(int)*size/3,hipMemcpyDeviceToHost,stream[2]);//Copying data from GPU to CPU

	hipEventRecord(stop);//Time is stopped
  hipEventSynchronize(stop);//The program waits here until all the operations of the events completed
	hipEventElapsedTime(&totaltime, start, stop);//The execution time is calculated
	printf("Total Execution Time = %f ms\n",totaltime);
	printf("C[size-1] = %d\n",C_Host[size-1]);

	hipHostFree(A_Host);//Array on the memory of the CPU is freed
	hipHostFree(B_Host);//Array on the memory of the CPU is freed
	hipHostFree(C_Host);//Array on the memory of the CPU is freed

	hipFree(A1_GPU);//Array on the memory of the GPU is freed
	hipFree(B1_GPU);//Array on the memory of the GPU is freed
	hipFree(C1_GPU);//Array on the memory of the GPU is freed

	hipFree(A2_GPU);//Array on the memory of the GPU is freed
	hipFree(B2_GPU);//Array on the memory of the GPU is freed
	hipFree(C2_GPU);//Array on the memory of the GPU is freed

	hipFree(A3_GPU);//Array on the memory of the GPU is freed
	hipFree(B3_GPU);//Array on the memory of the GPU is freed
	hipFree(C3_GPU);//Array on the memory of the GPU is freed

	hipError_t err = hipGetLastError();//Catchs the latest error occured on the GPU
	if ( err != hipSuccess )
		printf("CUDA Error: %s\n",hipGetErrorString(err));
}
